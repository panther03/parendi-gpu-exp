#include "hip/hip_runtime.h"
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
// Primary header is compatible with pre-C++11, collective algorithm headers require C++11
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <stdio.h>
#include <cuda/barrier>

#define ITERS 1000000
#define THREADS_PER_BLOCK 256

//__device__ cuda::barrier<cuda::thread_scope_system> B(TOTAL_THREADS);
namespace cg = cooperative_groups;

__global__ void xorShift(int *R, int fpt) {
    int x = threadIdx.x;
    int y = blockIdx.x;
    int z = threadIdx.x + 1;
    cg::grid_group g = cg::this_grid();
    volatile int w = blockDim.x - threadIdx.x;
    for (int j = 0; j < ITERS; j++) {
        for (int i = 0; i < fpt; i++) {
            int tmp=(x^(x<<15)); x=y; y=z; z=w;  
            w=(w^(w>>21))^(tmp^(tmp>>4)); 
        }
        //B.arrive_and_wait();
        g.sync();
    }
    //R[blockIdx.x * blockDim.x + threadIdx.x] = w;
} 

__global__ void xorShiftNoSync(int *R, int fpt) {
    int x = threadIdx.x;
    int y = blockIdx.x;
    int z = threadIdx.x + 1;
    volatile int w = blockDim.x - threadIdx.x;
    for (int j = 0; j < ITERS; j++) {
        for (int i = 0; i < fpt; i++) {
            int tmp=(x^(x<<15)); x=y; y=z; z=w;  
            w=(w^(w>>21))^(tmp^(tmp>>4)); 
        }
    }
    //R[blockIdx.x * blockDim.x + threadIdx.x] = w;
} 

static inline void launchKernel(void* kernelFunc, int *d_R, int fpt, int tb, int tpb) {
    dim3 gridDim(tb);
    dim3 blockDim(tpb);
    void *args[] = {(void*)&d_R, (void*)&fpt};
    hipLaunchCooperativeKernel(reinterpret_cast<const void*>(kernelFunc), gridDim, blockDim, args);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
}

int main(int argc, char **argv)
{
    assert((argc == 3) && "Need number of fibers and number of threads.");

    int fpt = atoi(argv[1]);
    int tt = atoi(argv[2]);
    std::cerr << "# fibers per thread: " << fpt << "\n# threads: " << tt << std::endl;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = 1 * sizeof(int);

    // Allocate the device output vector C
    int *d_R = NULL;
    err        = hipMalloc((void **)&d_R, size);

    // Allocate the host input vector A
    //int *h_R = (int *)malloc(size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int tb = tt/THREADS_PER_BLOCK;
    int tpb = THREADS_PER_BLOCK;
    
    if (tb == 0) {
        tb = 1;
        tpb = tt;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float sync_ms = 0;
    float nosync_ms = 0;


    /*
    hipEventRecord(start);
    launchKernel((void*)xorShift, d_R, fpt, tb, tpb);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sync_ms, start, stop);
    */
    
    hipEventRecord(start);
    launchKernel((void*)xorShiftNoSync, d_R, fpt, tb, tpb);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&nosync_ms, start, stop);


    std::cout << nosync_ms << "," << sync_ms << std::endl;
    // print 'efficiency'
    //std::cout << (nosync_ms)/(sync_ms) * 100. << std::endl;

    err = hipFree(d_R);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector R (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    //free(h_R);
    return 0;
}
